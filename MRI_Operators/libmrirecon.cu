#include "hip/hip_runtime.h"
// *----------------------------------------------
// 	Author Contact Information:
// 	Jiulong Liu
// 	matliuj@nus.edu.sg || jiu.liu@gmail.com
// 	Department of Mathematics, National University of Singapore

//
// If you find this code useful, you may cite the following reference:
//  



#include <math.h>
#include <malloc.h>
#include <hipfft/hipfft.h>
#include <hipblas.h>
#include <hipblas.h>
//#include <hipfft/hipfftXt.h>
#define ABS(a) (a>0?a:-(a))

#define BLOCK_SIZE_x 16
#define BLOCK_SIZE_y 16

extern "C" void fft2d(float *ur, float *ui, float *fr, float *fi, int N);
extern "C" void ifft2d(float *ur, float *ui, float *fr, float *fi, int N);
extern "C" void Ax_mri(float *ur, float *ui, float *fr, float *fi, float *mask,int N );
extern "C" void Atx_mri(float *ur, float *ui, float *fr, float *fi, float *mask, int N );
extern "C" void conjugate_grad(float *br, float *bi, float *xr, float *xi,  float mu, float rho, int maxit, float CG_tol0, float *mask, int N );
extern "C" void mrirecon_conjugate_grad(float *xr, float *yr, float *yi, float *xpr,  float mu, float rho, int CG_maxit, float CG_tol0, float *mask,int N, float lambda);
extern "C" void mrirecon_admm(float *xr, float *yr, float *yi, float *xpr,  float mu, float rho, int CG_maxit, float CG_tol0, int N_iter, float *mask,int N, float lambda);

__global__ void real2complex(float *fr, float *fi, hipfftComplex *fc, int N)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = j*N + i;
	if (i<N && j<N)
	{
		fc[index].x = fr[index];
		fc[index].y = fi[index];
	}
}

__global__ void complex2real(hipfftComplex *fc, float *fr, float *fi,int N)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = j*N + i;
	if (i<N && j<N)
	{
		fr[index] = fc[index].x;
		fi[index] = fc[index].y;
	}
}

__global__ void complex2real_scale(hipfftComplex *fc, float *fr, float *fi,int N)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = j*N + i;
	if (i<N && j<N)
	{
		fr[index] = fc[index].x/(float)N;//((float)N*(float)N);
		fi[index] = fc[index].y/(float)N;//((float)N*(float)N);
		//divide by number of elements to recover value
	}
}


__global__ void complex2magtitude(hipfftComplex *fc, float *fr, int N)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = j*N + i;
	if (i<N && j<N)
	{
		fr[index] = sqrt(fc[index].x*fc[index].x+fc[index].y*fc[index].y);
		
	}
}



__global__ void kspacedownsample(hipfftComplex *fc, float *mask,int N)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = threadIdx.y + blockIdx.y*blockDim.y;
	int index = j*N + i;
	if (i<N && j<N)
	{
		fc[index].x = fc[index].x*mask[index];
		fc[index].y = fc[index].y*mask[index];
		
	}
}




void fft2d(float *ur, float *ui, float *fr, float *fi, int N )
{   


//	N=192;
	float  *fr_d, *fi_d, *ur_d, *ui_d;//*k_d,

	hipMalloc((void**)&fr_d, sizeof(float)*N*N);
	hipMalloc((void**)&fi_d, sizeof(float)*N*N);
	hipMalloc((void**)&ur_d, sizeof(float)*N*N);
	hipMalloc((void**)&ui_d, sizeof(float)*N*N);

	hipMemcpy(fr_d, fr, sizeof(float)*N*N, hipMemcpyHostToDevice);
	hipMemcpy(fi_d, fi, sizeof(float)*N*N, hipMemcpyHostToDevice);
	hipfftComplex *f_fft_dc, *f_dc;
	hipMalloc((void**)&f_fft_dc, sizeof(hipfftComplex)*N*N);
	hipMalloc((void**)&f_dc, sizeof(hipfftComplex)*N*N);
	dim3 dimBlock (BLOCK_SIZE_x, BLOCK_SIZE_y);
	dim3 dimGrid_t((N+dimBlock.x-1)/dimBlock.x,(N+dimBlock.y-1)/dimBlock.y);
	real2complex<<<dimGrid_t, dimBlock>>>(fr_d, fi_d, f_dc, N);
	hipfftHandle plan;
	hipfftPlan2d(&plan, N, N, HIPFFT_C2C);

	hipfftExecC2C(plan, f_dc, f_fft_dc , HIPFFT_FORWARD);
//	hipfftExecC2C(plan, f_fft_dc, f_dc, HIPFFT_BACKWARD);
	complex2real_scale<<<dimGrid_t, dimBlock>>>(f_fft_dc, ur_d, ui_d, N);
//	complex2real<<<dimGrid_t, dimBlock>>>(f_dc, ur_d, ui_d, N);
	hipMemcpy(ur, ur_d, sizeof(float)*N*N, hipMemcpyDeviceToHost);
	hipMemcpy(ui, ui_d, sizeof(float)*N*N, hipMemcpyDeviceToHost);

	hipfftDestroy(plan);
	hipFree(fr_d);
	hipFree(fi_d);
	hipFree(ur_d);
	hipFree(ui_d);
	hipFree(f_fft_dc);
	hipFree(f_dc);
}



void ifft2d(float *ur, float *ui, float *fr, float *fi, int N )
{   


//	N=192;
	float  *fr_d, *fi_d, *ur_d, *ui_d;//*k_d,

	hipMalloc((void**)&fr_d, sizeof(float)*N*N);
	hipMalloc((void**)&fi_d, sizeof(float)*N*N);
	hipMalloc((void**)&ur_d, sizeof(float)*N*N);
	hipMalloc((void**)&ui_d, sizeof(float)*N*N);

	hipMemcpy(fr_d, fr, sizeof(float)*N*N, hipMemcpyHostToDevice);
	hipMemcpy(fi_d, fi, sizeof(float)*N*N, hipMemcpyHostToDevice);
	hipfftComplex *f_ifft_dc, *f_dc;
	hipMalloc((void**)&f_ifft_dc, sizeof(hipfftComplex)*N*N);
	hipMalloc((void**)&f_dc, sizeof(hipfftComplex)*N*N);
	dim3 dimBlock (BLOCK_SIZE_x, BLOCK_SIZE_y);
	dim3 dimGrid_t((N+dimBlock.x-1)/dimBlock.x,(N+dimBlock.y-1)/dimBlock.y);
	real2complex<<<dimGrid_t, dimBlock>>>(fr_d, fi_d, f_dc, N);
	hipfftHandle plan;
	hipfftPlan2d(&plan, N, N, HIPFFT_C2C);

//	hipfftExecC2C(plan, f_dc, f_ifft_dc , HIPFFT_FORWARD);
	hipfftExecC2C(plan, f_dc, f_ifft_dc, HIPFFT_BACKWARD);
	complex2real_scale<<<dimGrid_t, dimBlock>>>(f_ifft_dc, ur_d, ui_d, N);
	hipMemcpy(ur, ur_d, sizeof(float)*N*N, hipMemcpyDeviceToHost);
	hipMemcpy(ui, ui_d, sizeof(float)*N*N, hipMemcpyDeviceToHost);
	hipfftDestroy(plan);
	hipFree(fr_d);
	hipFree(fi_d);
	hipFree(ur_d);
	hipFree(ui_d);
	hipFree(f_ifft_dc);
	hipFree(f_dc);

}


void Ax_mri(float *ur, float *ui, float *fr, float *fi, float *mask,int N )
{   


//	N=192;
	float  *fr_d, *fi_d, *ur_d, *ui_d, *mask_d;//*k_d,

	hipMalloc((void**)&fr_d, sizeof(float)*N*N);
	hipMalloc((void**)&fi_d, sizeof(float)*N*N);
	hipMalloc((void**)&ur_d, sizeof(float)*N*N);
	hipMalloc((void**)&ui_d, sizeof(float)*N*N);
	hipMalloc((void**)&mask_d, sizeof(float)*N*N);
	hipMemcpy(fr_d, fr, sizeof(float)*N*N, hipMemcpyHostToDevice);
	hipMemcpy(fi_d, fi, sizeof(float)*N*N, hipMemcpyHostToDevice);
	hipMemcpy(mask_d, mask, sizeof(float)*N*N, hipMemcpyHostToDevice);
	hipfftComplex *f_fft_dc, *f_dc;
	hipMalloc((void**)&f_fft_dc, sizeof(hipfftComplex)*N*N);
	hipMalloc((void**)&f_dc, sizeof(hipfftComplex)*N*N);
	dim3 dimBlock (BLOCK_SIZE_x, BLOCK_SIZE_y);
	dim3 dimGrid_t((N+dimBlock.x-1)/dimBlock.x,(N+dimBlock.y-1)/dimBlock.y);
	real2complex<<<dimGrid_t, dimBlock>>>(fr_d, fi_d, f_dc, N);
	hipfftHandle plan;
	hipfftPlan2d(&plan, N, N, HIPFFT_C2C);

	hipfftExecC2C(plan, f_dc, f_fft_dc , HIPFFT_FORWARD);
	kspacedownsample<<<dimGrid_t, dimBlock>>>(f_fft_dc, mask_d, N);
	complex2real_scale<<<dimGrid_t, dimBlock>>>(f_fft_dc, ur_d, ui_d, N);

	hipMemcpy(ur, ur_d, sizeof(float)*N*N, hipMemcpyDeviceToHost);
	hipMemcpy(ui, ui_d, sizeof(float)*N*N, hipMemcpyDeviceToHost);
	hipfftDestroy(plan);
	hipFree(fr_d);
	hipFree(fi_d);
	hipFree(ur_d);
	hipFree(ui_d);
	hipFree(f_fft_dc);
	hipFree(f_dc);
}


void Atx_mri(float *ur, float *ui, float *fr, float *fi, float *mask, int N )
{   


//	N=192;
	float  *fr_d, *fi_d, *ur_d, *ui_d, *mask_d;//*k_d,

	hipMalloc((void**)&fr_d, sizeof(float)*N*N);
	hipMalloc((void**)&fi_d, sizeof(float)*N*N);
	hipMalloc((void**)&ur_d, sizeof(float)*N*N);
	hipMalloc((void**)&ui_d, sizeof(float)*N*N);
	hipMalloc((void**)&mask_d, sizeof(float)*N*N);
	hipMemcpy(fr_d, fr, sizeof(float)*N*N, hipMemcpyHostToDevice);
	hipMemcpy(fi_d, fi, sizeof(float)*N*N, hipMemcpyHostToDevice);
	hipMemcpy(mask_d, mask, sizeof(float)*N*N, hipMemcpyHostToDevice);
	hipfftComplex *f_ifft_dc, *f_dc;
	hipMalloc((void**)&f_ifft_dc, sizeof(hipfftComplex)*N*N);
	hipMalloc((void**)&f_dc, sizeof(hipfftComplex)*N*N);
	dim3 dimBlock (BLOCK_SIZE_x, BLOCK_SIZE_y);
	dim3 dimGrid_t((N+dimBlock.x-1)/dimBlock.x,(N+dimBlock.y-1)/dimBlock.y);
	real2complex<<<dimGrid_t, dimBlock>>>(fr_d, fi_d, f_dc, N);
	kspacedownsample<<<dimGrid_t, dimBlock>>>(f_dc, mask_d, N);
	hipfftHandle plan;
	hipfftPlan2d(&plan, N, N, HIPFFT_C2C);

//	hipfftExecC2C(plan, f_dc, f_ifft_dc , HIPFFT_FORWARD);
	hipfftExecC2C(plan, f_dc, f_ifft_dc, HIPFFT_BACKWARD);
	complex2real_scale<<<dimGrid_t, dimBlock>>>(f_ifft_dc, ur_d, ui_d, N);
	hipMemcpy(ur, ur_d, sizeof(float)*N*N, hipMemcpyDeviceToHost);
	hipMemcpy(ui, ui_d, sizeof(float)*N*N, hipMemcpyDeviceToHost);
	hipfftDestroy(plan);
	hipFree(fr_d);
	hipFree(fi_d);
	hipFree(ur_d);
	hipFree(ui_d);
	hipFree(f_ifft_dc);
	hipFree(f_dc);
}







__global__ void wx_2d_kernel_c(hipfftComplex *wx, hipfftComplex *x,int nx,int ny)
{	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int ix=bx*BLOCK_SIZE_x+tx;
	int iy=by*BLOCK_SIZE_y+ty;
    if(ix<nx&&iy<ny)
	{	int idx;
		idx=iy*nx+ix;
		if(ix<nx-1)		
		{
			wx[2 * idx].x = x[idx].x - x[idx + 1].x; // check
			wx[2 * idx].y = x[idx].y - x[idx + 1].y;
		}
		else
		{
			wx[2 * idx].x = 0.0f;
			wx[2 * idx].y = 0.0f;
		}
		if(iy<ny-1)		
		{

			wx[2 * idx + 1].x = x[idx].x - x[idx + nx].x;
			wx[2 * idx + 1].y = x[idx].y - x[idx + nx].y;
		}
		else
		{
			wx[2 * idx + 1].x = 0.0f;
			wx[2 * idx + 1].y = 0.0f;
		}
	}
}
void wx_2d_dc(hipfftComplex *wx_d, hipfftComplex *x_d,int nx,int ny)
{   dim3 dimBlock(BLOCK_SIZE_x,BLOCK_SIZE_y);
	dim3 dimGrid_t((nx+dimBlock.x-1)/dimBlock.x,(ny+dimBlock.y-1)/dimBlock.y);
	wx_2d_kernel_c<<<dimGrid_t, dimBlock>>>(wx_d,x_d,nx,ny);
	hipDeviceSynchronize();			
}

__global__ void wtx_2d_kernel_c(hipfftComplex *x, hipfftComplex *wx,int nx,int ny)
{	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int ix=bx*BLOCK_SIZE_x+tx;
	int iy=by*BLOCK_SIZE_y+ty;
    if(ix<nx&&iy<ny)
	{	
		int idx;
		idx=iy*nx+ix;
		x[idx].x = 0.0f;
		x[idx].y = 0.0f;
		if(ix==0)		
		{
			x[idx].x+=wx[2*idx].x;
			x[idx].y+=wx[2*idx].y;

		}
		else
		{	if(ix<nx-1)
			{
				x[idx].x+=-wx[2*(idx-1)].x+wx[2*idx].x;
				x[idx].y+=-wx[2*(idx-1)].y+wx[2*idx].y;
			}
			else
			{
				x[idx].x+=-wx[2*(idx-1)].x;
				x[idx].y+=-wx[2*(idx-1)].y;
			}
		}
		if(iy==0)		
		{
			x[idx].x+=wx[2*idx+1].x;
			x[idx].y+=wx[2*idx+1].y;
		}
		else
		{	if(iy<ny-1)
			{
				x[idx].x+=-wx[2*(idx-nx)+1].x+wx[2*idx+1].x;
				x[idx].y+=-wx[2*(idx-nx)+1].y+wx[2*idx+1].y;
			}
			else
			{
				x[idx].x+=-wx[2*(idx-nx)+1].x;
				x[idx].y+=-wx[2*(idx-nx)+1].y;
			}
		}
		
	}
}
void wtx_2d_dc(hipfftComplex *x_d, hipfftComplex *wx_d,int nx,int ny)
{   dim3 dimBlock(BLOCK_SIZE_x,BLOCK_SIZE_y);
	dim3 dimGrid_t((nx+dimBlock.x-1)/dimBlock.x,(ny+dimBlock.y-1)/dimBlock.y);			        
	wtx_2d_kernel_c<<<dimGrid_t, dimBlock>>>(x_d,wx_d,nx,ny);
	hipDeviceSynchronize();			
}

__global__ void shrink_2d_kernel_c(hipfftComplex *y, hipfftComplex *x,float s,int nx,int ny)
{	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int ix=bx*BLOCK_SIZE_x+tx;
	int iy=by*BLOCK_SIZE_y+ty;
    if(ix<nx&&iy<ny)
	{	int idx;
		float tmp,d;
		idx=2*(iy*nx+ix);		
		tmp=(float)sqrt(x[idx].x*x[idx].x-x[idx].y*x[idx].y+x[idx+1].x*x[idx+1].x-x[idx+1].y*x[idx+1].y);
        if(tmp>s)
        {
			d=(tmp-s)/tmp;
			y[idx].x=x[idx].x*d;
			y[idx].y=x[idx].y*d;
			y[idx+1].x=x[idx+1].x*d;
			y[idx+1].y=x[idx+1].y*d;
		}
        else
        {
			y[idx].x=0.0f;
			y[idx].y=0.0f;
			y[idx+1].x=0.0f;
			y[idx+1].y=0.0f;
		}
	}
}
void shrink_2d_dc(hipfftComplex *y, hipfftComplex *x,float s,int nx,int ny)
{   
	dim3 dimBlock(BLOCK_SIZE_x,BLOCK_SIZE_y);
	dim3 dimGrid_t((nx+dimBlock.x-1)/dimBlock.x,(ny+dimBlock.y-1)/dimBlock.y);
	shrink_2d_kernel_c<<<dimGrid_t, dimBlock>>>(y,x,s,nx,ny);
	hipDeviceSynchronize();			
}

__global__ void mulC_kernel_c(hipfftComplex *x, float c,int nx,int ny)
{	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int ix=bx*BLOCK_SIZE_x+tx;
	int iy=by*BLOCK_SIZE_y+ty;
    if(ix<nx&&iy<ny)
	{	int idx=(iy*nx+ix);		
		x[idx].x *= c;
		x[idx].y *= c;
	}
}
__global__ void mulC_plus_kernel_c(hipfftComplex *y, hipfftComplex *x,float s,int nx,int ny)
{	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int ix=bx*BLOCK_SIZE_x+tx;
	int iy=by*BLOCK_SIZE_y+ty;
    if(ix<nx&&iy<ny)
	{	int idx=(iy*nx+ix);		
		y[idx].x += s*x[idx].x;
		y[idx].y += s*x[idx].y;
	}
}

__global__ void plus_kernel_c(hipfftComplex *y, hipfftComplex *x,int nx,int ny)
{	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int ix=bx*BLOCK_SIZE_x+tx;
	int iy=by*BLOCK_SIZE_y+ty;
    if(ix<nx&&iy<ny)
	{	int idx=(iy*nx+ix);		
		y[idx].x += x[idx].x;
		y[idx].y += x[idx].y;
	}
}

__global__ void addition_kernel_c(hipfftComplex *z,hipfftComplex *y, hipfftComplex *x,int nx,int ny)
{	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int ix=bx*BLOCK_SIZE_x+tx;
	int iy=by*BLOCK_SIZE_y+ty;
    if(ix<nx&&iy<ny)
	{	int idx=(iy*nx+ix);		
		z[idx].x = y[idx].x + x[idx].x;
		z[idx].y = y[idx].y + x[idx].y;
	}
}


__global__ void subtraction_kernel_c(hipfftComplex *z,hipfftComplex *y, hipfftComplex *x,int nx,int ny)
{	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int ix=bx*BLOCK_SIZE_x+tx;
	int iy=by*BLOCK_SIZE_y+ty;
    if(ix<nx&&iy<ny)
	{	int idx=(iy*nx+ix);		
		z[idx].x = y[idx].x - x[idx].x;
		z[idx].y = y[idx].y - x[idx].y;
	}
}


__global__ void minus_kernel_c(hipfftComplex *y, hipfftComplex *x,int nx,int ny)
{	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int ix=bx*BLOCK_SIZE_x+tx;
	int iy=by*BLOCK_SIZE_y+ty;
    if(ix<nx&&iy<ny)
	{	int idx=(iy*nx+ix);		
		y[idx].x -= x[idx].x;
		y[idx].y -= x[idx].y;
	}
}


__global__ void initialize_kernel_c(hipfftComplex *y,int nx,int ny)
{	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int ix=bx*BLOCK_SIZE_x+tx;
	int iy=by*BLOCK_SIZE_y+ty;
    if(ix<nx&&iy<ny)
	{	int idx=(iy*nx+ix);		
		y[idx].x = 0.0f;
		y[idx].y = 0.0f;
	}
}

__global__ void initialize_kernel(float *y,int nx,int ny)
{	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int ix=bx*BLOCK_SIZE_x+tx;
	int iy=by*BLOCK_SIZE_y+ty;
    if(ix<nx&&iy<ny)
	{	int idx=(iy*nx+ix);		
		y[idx]= 0.0f;
	}
}
__global__ void equal_kernel_c(hipfftComplex *y, hipfftComplex *x,int nx,int ny)
{	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int ix=bx*BLOCK_SIZE_x+tx;
	int iy=by*BLOCK_SIZE_y+ty;
    if(ix<nx&&iy<ny)
	{	int idx=(iy*nx+ix);		
		y[idx].x = x[idx].x;
		y[idx].y = x[idx].y;
	}
}


__global__ void equalC_kernel_c(hipfftComplex *y, float c,int nx,int ny)
{	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int ix=bx*BLOCK_SIZE_x+tx;
	int iy=by*BLOCK_SIZE_y+ty;
    if(ix<nx&&iy<ny)
	{	int idx=(iy*nx+ix);		
		y[idx].x = c;
		y[idx].y = c+0.10f;
	}
}
__global__ void equalf_kernel_c(hipfftComplex *y, float *c,int nx,int ny)
{	int bx=blockIdx.x;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int ix=bx*BLOCK_SIZE_x+tx;
	int iy=by*BLOCK_SIZE_y+ty;
    if(ix<nx&&iy<ny)
	{	int idx=(iy*nx+ix);		
		y[idx].x = c[idx];
		y[idx].y = c[idx]+0.10f;
	}
}
void Ax_mri_d(hipfftComplex *fc,hipfftComplex *uc, float *mask_d,int N, hipfftHandle plan)
{   



	dim3 dimBlock (BLOCK_SIZE_x, BLOCK_SIZE_y);
	dim3 dimGrid_t((N+dimBlock.x-1)/dimBlock.x,(N+dimBlock.y-1)/dimBlock.y);
//	hipfftHandle plan;
//	hipfftPlan2d(&plan, N, N, HIPFFT_C2C);

	hipfftExecC2C(plan, uc, fc , HIPFFT_FORWARD);
	mulC_kernel_c<<<dimGrid_t, dimBlock>>>(fc, 1.0f/((float)N), N, N);
	kspacedownsample<<<dimGrid_t, dimBlock>>>(fc, mask_d, N);
//	hipfftDestroy(plan);
}



void Atx_mri_d(hipfftComplex *uc,hipfftComplex *fc, float *mask_d,int N, hipfftHandle plan)
{   

	dim3 dimBlock (BLOCK_SIZE_x, BLOCK_SIZE_y);
	dim3 dimGrid_t((N+dimBlock.x-1)/dimBlock.x,(N+dimBlock.y-1)/dimBlock.y);

//	hipfftHandle plan;
//	hipfftPlan2d(&plan, N, N, HIPFFT_C2C);

	kspacedownsample<<<dimGrid_t, dimBlock>>>(fc, mask_d, N);
	hipfftExecC2C(plan, fc, uc, HIPFFT_BACKWARD);
	mulC_kernel_c<<<dimGrid_t, dimBlock>>>(uc, 1.0f/((float)N), N, N);
//	hipfftDestroy(plan);
}

void conjugate_grad_dc(hipfftComplex *b_d, hipfftComplex *x_d,  float mu, float rho, int CG_maxit, float CG_tol0, float *mask_d,int N, hipfftComplex *tmpy, hipfftComplex *tmpatax, hipfftComplex *tmpwx,  hipfftComplex *tmpwtwx, hipfftComplex *r,hipfftComplex *p,hipfftComplex *tmpdot, float *tmpdotr, float *tmpdoti, hipfftHandle plan)
{

	int nx=N, ny=N, iter;	
	float rsold,rsnew,tmpdot_h[1],CG_tol,alpha,beta;
	dim3 dimBlock(BLOCK_SIZE_x,BLOCK_SIZE_y);
	dim3 dimGrid_x((nx+dimBlock.x-1)/dimBlock.x,(ny+dimBlock.y-1)/dimBlock.y);


	Ax_mri_d(tmpy,x_d, mask_d, N, plan);
	Atx_mri_d(tmpatax,tmpy, mask_d, N, plan);
	wx_2d_dc(tmpwx,x_d,nx,ny);
	wtx_2d_dc(tmpwtwx,tmpwx,nx,ny);	
	mulC_plus_kernel_c<<<dimGrid_x, dimBlock>>>(tmpatax,tmpwtwx,mu,nx,ny);hipDeviceSynchronize();
	mulC_plus_kernel_c<<<dimGrid_x, dimBlock>>>(tmpatax,x_d,rho,nx,ny);hipDeviceSynchronize();
	subtraction_kernel_c<<<dimGrid_x, dimBlock>>>(r,b_d,tmpatax,nx,ny);hipDeviceSynchronize();
	equal_kernel_c<<<dimGrid_x, dimBlock>>>(p,r,nx,ny);hipDeviceSynchronize();
	hipblasHandle_t h;
	hipblasCreate(&h);
	hipblasSetPointerMode(h, HIPBLAS_POINTER_MODE_DEVICE);
//	equalC_kernel_c<<<dimGrid_x, dimBlock>>>(r,(float)0.3,nx,ny);hipDeviceSynchronize(); //will be 0.25 * number of pixels
	hipblasCdotc(h,nx*ny,r,1,r,1,tmpdot);//tmpdot tmpdot_h2
	complex2real<<<dimGrid_x, dimBlock>>>(tmpdot, tmpdotr, tmpdoti, 1);
	hipMemcpy(tmpdot_h,tmpdotr,1*sizeof(float),hipMemcpyDeviceToHost);
	rsold=tmpdot_h[0];
	CG_tol=rsold*CG_tol0;
	
	for(iter=0; iter<CG_maxit && rsold>CG_tol;iter++) //&& rsold>CG_tol CG_maxit
	{
		Ax_mri_d(tmpy,p, mask_d, N, plan);
		Atx_mri_d(tmpatax,tmpy, mask_d, N, plan);
		wx_2d_dc(tmpwx,p,nx,ny);
		wtx_2d_dc(tmpwtwx,tmpwx,nx,ny);	

		mulC_plus_kernel_c<<<dimGrid_x, dimBlock>>>(tmpatax,tmpwtwx,mu,nx,ny);hipDeviceSynchronize();
		mulC_plus_kernel_c<<<dimGrid_x, dimBlock>>>(tmpatax,p,rho,nx,ny);hipDeviceSynchronize();
		hipblasCdotc(h,nx*ny,p,1,tmpatax,1,tmpdot);
		complex2real<<<dimGrid_x, dimBlock>>>(tmpdot, tmpdotr, tmpdoti, 1);
		hipMemcpy(tmpdot_h,tmpdotr,1*sizeof(float),hipMemcpyDeviceToHost);
		rsnew=tmpdot_h[0];
		if(rsnew<CG_tol) break;
		alpha=rsold/rsnew;
		mulC_plus_kernel_c<<<dimGrid_x, dimBlock>>>(x_d,p,alpha,nx,ny);hipDeviceSynchronize();
		mulC_plus_kernel_c<<<dimGrid_x, dimBlock>>>(r,tmpatax,-alpha,nx,ny);hipDeviceSynchronize();
		hipblasCdotc(h,nx*ny,r,1,r,1,tmpdot);
		complex2real<<<dimGrid_x, dimBlock>>>(tmpdot, tmpdotr, tmpdoti, 1);
		hipMemcpy(tmpdot_h,tmpdotr,1*sizeof(float),hipMemcpyDeviceToHost);
		rsnew=tmpdot_h[0];
		beta=rsnew/rsold;
		rsold=rsnew;      
		mulC_kernel_c<<<dimGrid_x, dimBlock>>>(p,beta,nx,ny);hipDeviceSynchronize();
		plus_kernel_c<<<dimGrid_x, dimBlock>>>(p,r,nx,ny);hipDeviceSynchronize();

	}

	hipblasDestroy(h);
//	equalC_kernel_c<<<dimGrid_x, dimBlock>>>(x_d,3.0,nx,ny);hipDeviceSynchronize();
//	equalf_kernel_c<<<dimGrid_x, dimBlock>>>(x_d,mask_d,nx,ny);hipDeviceSynchronize();
//	equal_kernel_c<<<dimGrid_x, dimBlock>>>(x_d,mask_d,nx,ny);hipDeviceSynchronize();

}

void conjugate_grad(float *br, float *bi, float *xr, float *xi,  float mu, float rho, int CG_maxit, float CG_tol0, float *mask, int N)
{

	
	//	N=192;
	float  *br_d, *bi_d, *xr_d, *xi_d, *mask_d;//*k_d,
	float *tmpdotr, *tmpdoti;
	hipfftComplex *tmpy, *tmpatax, *tmpwx,  *tmpwtwx, *r, *p, *tmpdot;
	hipfftHandle plan;
	hipfftPlan2d(&plan, N, N, HIPFFT_C2C);
	hipMalloc((void**)&tmpy, sizeof(hipfftComplex)*N*N);
	hipMalloc((void**)&tmpatax, sizeof(hipfftComplex)*N*N);
	hipMalloc((void**)&tmpwx, sizeof(hipfftComplex)*N*N*2);
	hipMalloc((void**)&tmpwtwx, sizeof(hipfftComplex)*N*N);
	hipMalloc((void**)&r, sizeof(hipfftComplex)*N*N);
	hipMalloc((void**)&p, sizeof(hipfftComplex)*N*N);
	hipMalloc(&tmpdot,1*sizeof(hipfftComplex));
	hipMalloc(&tmpdotr,1*sizeof(float));
	hipMalloc(&tmpdoti,1*sizeof(float));

	hipMalloc((void**)&br_d, sizeof(float)*N*N);
	hipMalloc((void**)&bi_d, sizeof(float)*N*N);
	hipMalloc((void**)&xr_d, sizeof(float)*N*N);
	hipMalloc((void**)&xi_d, sizeof(float)*N*N);
	hipMalloc((void**)&mask_d, sizeof(float)*N*N);
	hipMemcpy(br_d, br, sizeof(float)*N*N, hipMemcpyHostToDevice);
	hipMemcpy(bi_d, bi, sizeof(float)*N*N, hipMemcpyHostToDevice);
	hipMemcpy(xr_d, xr, sizeof(float)*N*N, hipMemcpyHostToDevice);
	hipMemcpy(xi_d, xi, sizeof(float)*N*N, hipMemcpyHostToDevice);
	hipMemcpy(mask_d, mask, sizeof(float)*N*N, hipMemcpyHostToDevice);
	hipfftComplex *b_dc, *x_dc;
	hipMalloc((void**)&b_dc, sizeof(hipfftComplex)*N*N);
	hipMalloc((void**)&x_dc, sizeof(hipfftComplex)*N*N);

	dim3 dimBlock (BLOCK_SIZE_x, BLOCK_SIZE_y);
	dim3 dimGrid_x((N+dimBlock.x-1)/dimBlock.x,(N+dimBlock.y-1)/dimBlock.y);
	real2complex<<<dimGrid_x, dimBlock>>>(br_d, bi_d, b_dc, N);
	real2complex<<<dimGrid_x, dimBlock>>>(xr_d, xi_d, x_dc, N);

	conjugate_grad_dc(b_dc, x_dc, mu, rho,  CG_maxit, CG_tol0, mask_d, N, tmpy, tmpatax, tmpwx,  tmpwtwx, r, p, tmpdot, tmpdotr, tmpdoti, plan);

	complex2real<<<dimGrid_x, dimBlock>>>(x_dc, xr_d, xi_d, N);

	hipMemcpy(xr, xr_d, sizeof(float)*N*N, hipMemcpyDeviceToHost);
	hipMemcpy(xi, xi_d, sizeof(float)*N*N, hipMemcpyDeviceToHost);
	hipfftDestroy(plan);
	hipFree(b_dc);
	hipFree(tmpy);
	hipFree(tmpatax);
	hipFree(tmpwx);
	hipFree(tmpwtwx);
	hipFree(r);
	hipFree(p);
	hipFree(tmpdot);
	hipFree(tmpdotr);
	hipFree(tmpdoti);

}







void mrirecon_conjugate_grad(float *xr, float *yr, float *yi, float *xpr,   float mu, float rho, int CG_maxit, float CG_tol0, float *mask,int N, float lambda)
{

	int nx=N,ny=N;



	hipfftComplex *tmpy,  *tmpatax,  *tmpwx,   *tmpwtwx,  *r, *p,  *tmpdot,  *x_dc, *y_dc, *xp_dc, *b_dc;
	float *tmpdotr,  *tmpdoti, *mask_d; 
	float *xr_d, *xi_d, *yr_d, *yi_d, *xpr_d, *xpi_d;
	hipMalloc((void**)&tmpy, sizeof(hipfftComplex)*N*N);
	hipMalloc((void**)&tmpatax, sizeof(hipfftComplex)*N*N);
	hipMalloc((void**)&tmpwx, sizeof(hipfftComplex)*N*N*2);
	hipMalloc((void**)&tmpwtwx, sizeof(hipfftComplex)*N*N);
	hipMalloc((void**)&r, sizeof(hipfftComplex)*N*N);
	hipMalloc((void**)&p, sizeof(hipfftComplex)*N*N);
	hipMalloc(&tmpdot,1*sizeof(hipfftComplex));
	hipMalloc(&tmpdotr,1*sizeof(float));
	hipMalloc(&tmpdoti,1*sizeof(float));
	hipMalloc((void**)&xr_d, sizeof(float)*N*N);
	hipMalloc((void**)&xi_d, sizeof(float)*N*N);
	hipMalloc((void**)&yr_d, sizeof(float)*N*N);
	hipMalloc((void**)&yi_d, sizeof(float)*N*N);
	hipMalloc((void**)&xpr_d, sizeof(float)*N*N);
	hipMalloc((void**)&xpi_d, sizeof(float)*N*N);
	hipMalloc((void**)&mask_d, sizeof(float)*N*N);
	hipMalloc((void**)&x_dc, sizeof(hipfftComplex)*N*N);
	hipMalloc((void**)&y_dc, sizeof(hipfftComplex)*N*N);
	hipMalloc((void**)&xp_dc, sizeof(hipfftComplex)*N*N);
	hipMalloc((void**)&b_dc, sizeof(hipfftComplex)*N*N);

	dim3 dimBlock(BLOCK_SIZE_x,BLOCK_SIZE_y);
	dim3 dimGrid_x((nx+dimBlock.x-1)/dimBlock.x,(ny+dimBlock.y-1)/dimBlock.y);
	dim3 dimGrid_wx((2*nx+dimBlock.x-1)/dimBlock.x,(ny+dimBlock.y-1)/dimBlock.y);
	hipfftHandle plan;
	hipfftPlan2d(&plan, N, N, HIPFFT_C2C);

	hipMemcpy(xr_d, xr, sizeof(float)*N*N, hipMemcpyHostToDevice);
	initialize_kernel<<<dimGrid_x, dimBlock>>>(xi_d,nx,ny);
//	hipMemcpy(xi_d, xi, sizeof(float)*N*N, hipMemcpyHostToDevice);
	hipMemcpy(yr_d, yr, sizeof(float)*N*N, hipMemcpyHostToDevice);
	hipMemcpy(yi_d, yi, sizeof(float)*N*N, hipMemcpyHostToDevice);
	hipMemcpy(xpr_d, xpr, sizeof(float)*N*N, hipMemcpyHostToDevice);
	initialize_kernel<<<dimGrid_x, dimBlock>>>(xpi_d,nx,ny);
//	hipMemcpy(xpi_d, xpi, sizeof(float)*N*N, hipMemcpyHostToDevice);
	hipMemcpy(mask_d, mask, sizeof(float)*N*N, hipMemcpyHostToDevice);


	real2complex<<<dimGrid_x, dimBlock>>>(xr_d, xi_d, x_dc, N);
	real2complex<<<dimGrid_x, dimBlock>>>(yr_d, yi_d, y_dc, N);
	real2complex<<<dimGrid_x, dimBlock>>>(xpr_d, xpi_d, xp_dc, N);
			
//	initialize_kernel_c<<<dimGrid_x, dimBlock>>>(x_dc,nx,ny);
	Atx_mri_d(b_dc, y_dc, mask_d, N, plan);
	wx_2d_dc(tmpwx,x_dc,nx,ny);// xp_dc for 
	wtx_2d_dc(tmpwtwx,tmpwx,nx,ny);//equal_kernel_c<<<dimGrid_x, dimBlock>>>(x_dc,b_dc,nx,ny);hipDeviceSynchronize();
	mulC_plus_kernel_c<<<dimGrid_x, dimBlock>>>(b_dc,tmpwtwx,mu,nx,ny);hipDeviceSynchronize();
	mulC_plus_kernel_c<<<dimGrid_x, dimBlock>>>(b_dc,xp_dc,rho,nx,ny);hipDeviceSynchronize();
	conjugate_grad_dc(b_dc, x_dc, mu, rho,  CG_maxit, CG_tol0, mask_d, N, tmpy, tmpatax, tmpwx,  tmpwtwx, r, p, tmpdot, tmpdotr, tmpdoti, plan);
	complex2magtitude<<<dimGrid_x, dimBlock>>>(x_dc, xr_d, N);	
// 	complex2real<<<dimGrid_x, dimBlock>>>(x_dc, xr_d, xi_d, N);

	hipMemcpy(xr, xr_d, sizeof(float)*N*N, hipMemcpyDeviceToHost);
//	hipMemcpy(xi, xi_d, sizeof(float)*N*N, hipMemcpyDeviceToHost);
  
	hipfftDestroy(plan);
	hipFree(tmpy);
	hipFree(tmpatax);
	hipFree(tmpwx);
	hipFree(tmpwtwx);
	hipFree(r);
	hipFree(p);
	hipFree(tmpdot);
	hipFree(tmpdotr);
	hipFree(tmpdoti);
	hipFree(xr_d);
	hipFree(xi_d);
	hipFree(yr_d);
	hipFree(yi_d);
	hipFree(xpr_d);
	hipFree(xpi_d);
	hipFree(mask_d);
	hipFree(x_dc);
	hipFree(y_dc);
	hipFree(xp_dc);
	hipFree(b_dc);

}






void mrirecon_admm_dc(hipfftComplex *x_d, hipfftComplex *y_d, hipfftComplex *xp_d, hipfftComplex *b_d,  float mu, float rho, int CG_maxit, float CG_tol0, int N_iter, float *mask_d,int N, float lambda, hipfftComplex *tmpy, hipfftComplex *tmpatax, hipfftComplex *tmpwx,  hipfftComplex *tmpwtwx, hipfftComplex *r,hipfftComplex *p, hipfftComplex *tmpdot, float *tmpdotr, float *tmpdoti, hipfftComplex *d_xs, hipfftComplex *v_xs)
{
	int nx=N,ny=N,n_iter;
	float lam=mu/lambda;

	hipfftHandle plan;
	hipfftPlan2d(&plan, N, N, HIPFFT_C2C);


		
	dim3 dimBlock(BLOCK_SIZE_x,BLOCK_SIZE_y);
	dim3 dimGrid_x((nx+dimBlock.x-1)/dimBlock.x,(ny+dimBlock.y-1)/dimBlock.y);
	dim3 dimGrid_wx((2*nx+dimBlock.x-1)/dimBlock.x,(ny+dimBlock.y-1)/dimBlock.y);
		
//	initialize_kernel_c<<<dimGrid_x, dimBlock>>>(x,nx,ny);
//	initialize_kernel_c<<<dimGrid_wx, dimBlock>>>(d_xs,2*nx,ny);
//	initialize_kernel_c<<<dimGrid_wx, dimBlock>>>(v_xs,2*nx,ny);



	    
	for(n_iter=0;n_iter<N_iter;n_iter++)
	{	
		Atx_mri_d(b_d, y_d, mask_d, N,plan);
		addition_kernel_c<<<dimGrid_wx, dimBlock>>>(tmpwx,d_xs,v_xs,2*nx,ny);hipDeviceSynchronize();
		wtx_2d_dc(tmpwtwx,tmpwx,nx,ny);
//		mulC_kernel<<<dimGrid_x, dimBlock>>>(tmpwtwx,mu,nx,ny);hipDeviceSynchronize();
		mulC_plus_kernel_c<<<dimGrid_x, dimBlock>>>(b_d,tmpwtwx,mu,nx,ny);hipDeviceSynchronize();
		mulC_plus_kernel_c<<<dimGrid_x, dimBlock>>>(b_d,xp_d,rho,nx,ny);hipDeviceSynchronize();
		conjugate_grad_dc(b_d, x_d, mu, rho,  CG_maxit, CG_tol0, mask_d, N, tmpy, tmpatax, tmpwx,  tmpwtwx, r, p, tmpdot, tmpdotr, tmpdoti,plan);
	    //conjugate_grad_dc(b_dc, x_dc, mu, rho,  CG_maxit, CG_tol0, mask_d, N, tmpy, tmpatax, tmpwx,  tmpwtwx, r, p, tmpdot, tmpdotr, tmpdoti);
		wx_2d_dc(tmpwx,x_d,nx,ny);
		minus_kernel_c<<<dimGrid_wx, dimBlock>>>(tmpwx,v_xs,2*nx,ny);hipDeviceSynchronize();
		shrink_2d_dc(d_xs,tmpwx,lam,nx,ny);
//		equal_kernel_c<<<dimGrid_wx, dimBlock>>>(d_xs,tmpwx,2*nx,ny);hipDeviceSynchronize();
		subtraction_kernel_c<<<dimGrid_wx, dimBlock>>>(v_xs,d_xs,tmpwx,2*nx,ny);hipDeviceSynchronize();

	}

	hipfftDestroy(plan);
//	initialize_kernel_c<<<dimGrid_x, dimBlock>>>(x_d,nx,ny);
	
}


void mrirecon_admm(float *xr, float *yr, float *yi, float *xpr,  float mu, float rho, int CG_maxit, float CG_tol0, int N_iter, float *mask,int N, float lambda)
{
//	float *tmpy,tmpdot_h[1],s,s2,lam,alpha,beta,CG_tol;
//	int n_iter,n_cg;
	int nx=N,ny=N;



	hipfftComplex *tmpy,  *tmpatax,  *tmpwx,   *tmpwtwx,  *r, *p,  *tmpdot, *d_xs, *v_xs, *x_dc, *y_dc, *xp_dc, *b_dc;
	float *tmpdotr,  *tmpdoti; 
	float *xr_d, *xi_d, *yr_d, *yi_d, *xpr_d, *xpi_d, *mask_d;
	hipMalloc((void**)&tmpy, sizeof(hipfftComplex)*N*N);
	hipMalloc((void**)&tmpatax, sizeof(hipfftComplex)*N*N);
	hipMalloc((void**)&tmpwx, sizeof(hipfftComplex)*N*N*2);
	hipMalloc((void**)&tmpwtwx, sizeof(hipfftComplex)*N*N);
	hipMalloc((void**)&r, sizeof(hipfftComplex)*N*N);
	hipMalloc((void**)&p, sizeof(hipfftComplex)*N*N);
	hipMalloc(&tmpdot,1*sizeof(hipfftComplex));
	hipMalloc(&tmpdotr,1*sizeof(float));
	hipMalloc(&tmpdoti,1*sizeof(float));
	hipMalloc((void**)&xr_d, sizeof(float)*N*N);
	hipMalloc((void**)&xi_d, sizeof(float)*N*N);
	hipMalloc((void**)&yr_d, sizeof(float)*N*N);
	hipMalloc((void**)&yi_d, sizeof(float)*N*N);
	hipMalloc((void**)&xpr_d, sizeof(float)*N*N);
	hipMalloc((void**)&xpi_d, sizeof(float)*N*N);
	hipMalloc((void**)&mask_d, sizeof(float)*N*N);
	hipMalloc((void**)&x_dc, sizeof(hipfftComplex)*N*N);
	hipMalloc((void**)&y_dc, sizeof(hipfftComplex)*N*N);
	hipMalloc((void**)&xp_dc, sizeof(hipfftComplex)*N*N);
	hipMalloc((void**)&b_dc, sizeof(hipfftComplex)*N*N);
	hipMalloc((void**)&d_xs, sizeof(hipfftComplex)*N*N*2);
	hipMalloc((void**)&v_xs, sizeof(hipfftComplex)*N*N*2);

	dim3 dimBlock(BLOCK_SIZE_x,BLOCK_SIZE_y);
	dim3 dimGrid_x((nx+dimBlock.x-1)/dimBlock.x,(ny+dimBlock.y-1)/dimBlock.y);
	dim3 dimGrid_wx((2*nx+dimBlock.x-1)/dimBlock.x,(ny+dimBlock.y-1)/dimBlock.y);
	hipMemcpy(xr_d, xr, sizeof(float)*N*N, hipMemcpyHostToDevice);
	initialize_kernel<<<dimGrid_x, dimBlock>>>(xi_d,nx,ny);
//	hipMemcpy(xi_d, xi, sizeof(float)*N*N, hipMemcpyHostToDevice);
	hipMemcpy(yr_d, yr, sizeof(float)*N*N, hipMemcpyHostToDevice);
	hipMemcpy(yi_d, yi, sizeof(float)*N*N, hipMemcpyHostToDevice);
	hipMemcpy(xpr_d, xpr, sizeof(float)*N*N, hipMemcpyHostToDevice);
	initialize_kernel<<<dimGrid_x, dimBlock>>>(xpi_d,nx,ny);
//	hipMemcpy(xpi_d, xpi, sizeof(float)*N*N, hipMemcpyHostToDevice);
	hipMemcpy(mask_d, mask, sizeof(float)*N*N, hipMemcpyHostToDevice);


	real2complex<<<dimGrid_x, dimBlock>>>(xr_d, xi_d, x_dc, N);
	real2complex<<<dimGrid_x, dimBlock>>>(yr_d, yi_d, y_dc, N);
	real2complex<<<dimGrid_x, dimBlock>>>(xpr_d, xpi_d, xp_dc, N);


	

		

	initialize_kernel_c<<<dimGrid_wx, dimBlock>>>(d_xs,2*nx,ny);
	initialize_kernel_c<<<dimGrid_wx, dimBlock>>>(v_xs,2*nx,ny);

	mrirecon_admm_dc(x_dc, y_dc, xp_dc, b_dc, mu, rho, CG_maxit, CG_tol0, N_iter, mask_d, N, lambda, tmpy, tmpatax, tmpwx, tmpwtwx, r, p, tmpdot, tmpdotr, tmpdoti, d_xs, v_xs);
	complex2magtitude<<<dimGrid_x, dimBlock>>>(x_dc, xr_d, N);
//	complex2real<<<dimGrid_x, dimBlock>>>(x_dc, xr_d, xi_d, N);
	hipMemcpy(xr, xr_d, sizeof(float)*N*N, hipMemcpyDeviceToHost);
//	hipMemcpy(xi, xi_d, sizeof(float)*N*N, hipMemcpyDeviceToHost);
   
	hipFree(tmpy);
	hipFree(tmpatax);
	hipFree(tmpwx);
	hipFree(tmpwtwx);
	hipFree(r);
	hipFree(p);
	hipFree(tmpdot);
	hipFree(tmpdotr);
	hipFree(tmpdoti);
	hipFree(xr_d);
	hipFree(xi_d);
	hipFree(yr_d);
	hipFree(yi_d);
	hipFree(xpr_d);
	hipFree(xpi_d);
	hipFree(mask_d);
	hipFree(x_dc);
	hipFree(y_dc);
	hipFree(xp_dc);
	hipFree(b_dc);
	hipFree(d_xs);
	hipFree(v_xs);



}




